#include "hip/hip_runtime.h"
// Microbenchmark to determine how many ports and/or banks in the register file

#include <iostream>

#define N (32 * 1024)
#define NUM_THDS (32 * 8)
#define NUM_REGS (N / NUM_THDS)

// Kernel function to add the elements of three arrays into fourth array
__global__
void add(float *x, float *y, float *z, float *a)
{
    float sums[NUM_REGS];

    for (int i = 0; i < NUM_REGS; ++i) {
        ++a[threadIdx.x + i * blockDim.x];
        int j = index + i * stride;
        sums[i] = a[j];
        sums[i] += x[j];
        sums[i] += y[j];
        sums[i] += z[j];
        a[j] = sums[i];
    }
}

int main(void)
{
    float *x, *y, *z, *a;

    // Allocate Unified Memory – accessible from CPU or GPU
    // FIXME: manually copy memory to and from device
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));
    hipMallocManaged(&z, N*sizeof(float));
    hipMallocManaged(&a, N*sizeof(float));

    // initialize arrays on the host
    for (int i = 0; i < N; i++) {
        a[i] = 100000.0f;
        x[i] = i * 1.0f;
        y[i] = i * 2.0f;
        z[i] = i * 3.0f;
    }

    // Run kernel on the GPU
    // P100 has 32 * 1024 32-bit registers in an SM
    // 128 registers per thread -> (32 * 1024) / 128 = 256 threads
    add<<<1, NUM_THDS>>>(x, y, z, a);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // print output
    //for (int i = 0; i < N; ++i)
    //    std::cout << a[i] << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);
    hipFree(z);
    hipFree(a);

    return 0;
}
